#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#define _USE_MATH_DEFINES

using namespace cooperative_groups;

// C++ interface
#define CHECK_CUDA(x) TORCH_CHECK(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)


torch::Tensor fourier_layer_cuda_forward(
		const torch::Tensor& head_q,
		const torch::Tensor& head_k,
		const torch::Tensor& paramR);

std::vector<torch::Tensor> fourier_layer_cuda_backward(
		const torch::Tensor& grad_Y,
		const torch::Tensor& head_q,
		const torch::Tensor& head_k,
		const torch::Tensor& paramR,
		const torch::Tensor& Y);


torch::Tensor fourier_layer_forward(
		const torch::Tensor& head_q,
		const torch::Tensor& head_k,
		const torch::Tensor& paramR) {
  
  CHECK_INPUT(head_q);
  CHECK_INPUT(head_k);
  CHECK_INPUT(paramR);
  //std::cout <<"done checking input."<<std::endl;
  const at::cuda::OptionalCUDAGuard device_guard(device_of(head_q));
  return fourier_layer_cuda_forward(head_q,head_k,paramR);
}

std::vector<torch::Tensor> fourier_layer_backward(
		const torch::Tensor& grad_Y,
		const torch::Tensor& head_q,
		const torch::Tensor& head_k,
		const torch::Tensor& paramR,
		const torch::Tensor& Y)  {
  CHECK_INPUT(grad_Y);
  CHECK_INPUT(head_q);
  CHECK_INPUT(head_k);
  CHECK_INPUT(paramR);
  CHECK_INPUT(Y);

  const at::cuda::OptionalCUDAGuard device_guard(device_of(head_q));
  return fourier_layer_cuda_backward(grad_Y, head_q, head_k, paramR, Y);

}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("forward" , &fourier_layer_forward , "FOURIER_LAYER forward  (CUDA)");
  m.def("backward", &fourier_layer_backward, "FOURIER_LAYER backward (CUDA)");
}



#if __CUDA_ARCH__ < 600
template <typename T>
__device__ double atomicAdd(T* address, T val)
{
    unsigned long long int* address_as_ull =
    (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
        __double_as_longlong(val +
        __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

namespace{
//const float eps = 1e-31f;

template <typename scalar_t>
__global__ void fourier_layer_cuda_forward_kernel(
          scalar_t* p_Y,
	const scalar_t* __restrict__ p_head_q,
    const scalar_t* __restrict__ p_head_k,
    const scalar_t* __restrict__ p_paramR,
	const size_t n_head,
	const size_t bsz, 
	const size_t qlen, 
	const size_t klen,
	const size_t d_head,
	const size_t N) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int b = blockIdx.y;

  if (i < N){
	  
	//extract n,b,q,k
	// size = [qlen, klen bsz, n_head]
	int q =   i / (klen*n_head);
	int k = ( i % (klen*n_head) ) / (n_head) ;
	int n =   i % n_head;
	
	scalar_t& result = p_Y[q*klen*bsz*n_head + k*bsz*n_head + b*n_head + n];
	//scalar_t& result = p_Y[i];
	
	const scalar_t* p_head_q_i = p_head_q + (q*bsz*n_head + b*n_head + n ) * d_head;
	const scalar_t* p_head_k_i = p_head_k + (k*bsz*n_head + b*n_head + n ) * d_head;
	result=1.0f;
	
	//sum on d
	scalar_t diff;
	for(int d=0; d<d_head; d++){	
	  //float diff = ( p_head_q[n,b,q,d] - p_head_k[n,b,k,d] ) 	 
	  diff = ( p_head_q_i[d] - p_head_k_i[d] ) * p_paramR[d] ;
				
	  if(abs(diff)<1e-30f) diff=1;
	  else diff = sinf(diff)/diff;
								
	  result = result * diff ;
	}
	
	//result *= __powf(p_paramR[0],d_head);	
	
  }//end if	
  
}

/*
template <typename scalar_t>
__device__ scalar_t reduce_sum(thread_group g, float *temp, scalar_t val){
    int lane = g.thread_rank();
    for (int i = g.size()/2; i > 0; i /= 2)    {
        temp[lane] = val;
        g.sync(); // wait for all threads to store
        if(lane<i) val += temp[lane + i];
		//g.sync();
		if(lane==0 && i%2==1 && i>2) val+=(temp[i-1]+temp[2*i-1]);
        g.sync(); // wait for all threads to load
    }
    return val; // note: only thread 0 will return full sum
}
*/

template <typename scalar_t>
__device__ scalar_t reduce_sum(thread_group g, float *temp, scalar_t val){
    int lane = g.thread_rank();
	temp[lane] = val;
	g.sync();
    for (int i = g.size()/2; i > 0; i /= 2)    {
		if( lane==0 && i%2==1 && i>2) temp[lane]+=(temp[i-1]+temp[2*i-1]);
		if(lane<i) temp[lane] += temp[lane + i];
		
        // wait for all threads to load
		g.sync();
    }
    return temp[0]; // note: only thread 0 will return full sum
}

// compute grad_head_q
template <typename scalar_t>
__global__ void fourier_layer_cuda_backward_kernel_q(
    const scalar_t* p_grad_Y,
	const scalar_t* __restrict__ p_head_q,
    const scalar_t* __restrict__ p_head_k,
    const scalar_t* __restrict__ p_paramR,
	const scalar_t* __restrict__ p_Y,
	      scalar_t* __restrict__ p_grad_head_q,
		  scalar_t* __restrict__ p_grad_head_k,
		  scalar_t* __restrict__ p_grad_paramR,
	const size_t n_head,
	const size_t bsz, 
	const size_t qlen, 
	const size_t klen,
	const size_t qleni,
	const size_t kleni,
	const size_t d_head,
	const size_t N_q) 
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int n = blockIdx.y;
  const int b = blockIdx.z;  

  if (i < N_q){
	// size(head_q) = [qlen, bsz, n_head, d_head]
	int ki =   i / (qleni*d_head) ;
	int qi = ( i % (qleni*d_head) ) / d_head;
	int d  =   i % d_head;	
	
	scalar_t* grad_head_q = p_grad_head_q + 0*bsz*n_head*d_head + b*n_head*d_head + n*d_head + d;
	scalar_t* grad_head_k = p_grad_head_k + 0*bsz*n_head*d_head + b*n_head*d_head + n*d_head + d;
	
	const scalar_t* p_head_q_i = p_head_q + 0*bsz*n_head*d_head + b*n_head*d_head + n*d_head + d;
	const scalar_t* p_head_k_i = p_head_k + 0*bsz*n_head*d_head + b*n_head*d_head + n*d_head + d;
	const scalar_t* p_Y_i      = p_Y      + 0*klen*bsz*n_head + 0*bsz*n_head + b*n_head + n;
	const scalar_t* p_grad_Y_i = p_grad_Y + 0*klen*bsz*n_head + 0*bsz*n_head + b*n_head + n;
	//sum on k
	scalar_t diff, temp,  grad_paramR=0;
	//for(int n=0; n<n_head; n++){	
	for( int qii=qi; qii<qlen; qii+=qleni){
	  scalar_t sum_k = 0;
	  for( int kii=ki; kii<klen; kii+=kleni){
	
	    diff = ( p_head_q_i[qii*bsz*n_head*d_head] - p_head_k_i[kii*bsz*n_head*d_head] ) *p_paramR[d];
				
	    if(abs(diff)<1.0e-30f) temp=0;
	    else temp =  1.0f/tanf(diff) - 1.0f/diff  ;
	  
	    temp *= p_Y_i[qii*klen*bsz*n_head + kii*bsz*n_head] * p_grad_Y_i[qii*klen*bsz*n_head + kii*bsz*n_head] * p_paramR[d] ;

	    atomicAdd( grad_head_k + kii*bsz*n_head*d_head , -temp );
	  
	    sum_k += temp;

	    grad_paramR += temp*diff ;	  
	  }
	  atomicAdd( grad_head_q + qii*bsz*n_head*d_head ,  sum_k );

	}
	atomicAdd( &p_grad_paramR[d], grad_paramR/(p_paramR[d]*p_paramR[d]) );
  }
  
}

}//end namespace
  
  
// 1. forward  
//template <typename T>
torch::Tensor fourier_layer_cuda_forward(
		const torch::Tensor& head_q,
		const torch::Tensor& head_k,
		const torch::Tensor& paramR)
{ 
  const auto n_head     = head_q.size(2);
  const auto bsz        = head_q.size(1);
  const auto qlen       = head_q.size(0);
  const auto klen       = head_k.size(0);
  const auto d_head     = head_k.size(3);
  
  const auto N = qlen* klen  * n_head;
  
  const int threads = 1024;
  const dim3 blocks((N + threads - 1) / threads, bsz);
  //const int blocks =  (N + threads - 1) / threads ;
  auto dev = head_q.get_device();
  auto options = torch::TensorOptions().dtype(head_q.dtype())
                                       .layout(torch::kStrided)
                                       .device(torch::kCUDA, dev)
                                       .requires_grad(true);
	
  auto Y = torch::zeros( {qlen, klen, bsz, n_head}, options );   
  
  
  AT_DISPATCH_FLOATING_TYPES(head_q.type(), "fourier_layer_forward_cuda", ([&] {
    fourier_layer_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
  Y.data<scalar_t>(), 
  head_q.data<scalar_t>(), 
  head_k.data<scalar_t>(), 
  paramR.data<scalar_t>(), 
  n_head, bsz, qlen, klen, d_head, N);
  }));
  //printf("Error in cuda: %s\n", hipGetLastError());

  return Y;
}


std::vector<torch::Tensor> fourier_layer_cuda_backward(
		const torch::Tensor& grad_Y,
		const torch::Tensor& head_q,
		const torch::Tensor& head_k,
		const torch::Tensor& paramR,
		const torch::Tensor& Y)
{
  int n_head     = head_q.size(2);
  int bsz        = head_q.size(1);
  int qlen       = head_q.size(0);
  int klen       = head_k.size(0);
  int d_head     = head_k.size(3);
  
  int stride = 16;
  int qleni = stride, kleni = stride;
  
  const int threads = qleni*kleni;
  auto const N_q = qleni*d_head*kleni;
  const dim3 blocks_q ( (N_q + threads - 1)/threads, n_head, bsz );
	
  auto grad_head_q = torch::zeros_like(head_q); //[qlen,bsz,n_head,d_head]
  auto grad_head_k = torch::zeros_like(head_k); //[klen,bsz,n_head,d_head]
  auto grad_paramR = torch::zeros_like(paramR); //[1]
  
  AT_DISPATCH_FLOATING_TYPES(head_q.type(), "fourier_layer_cuda_backward_kernel_q", 
	([&] {fourier_layer_cuda_backward_kernel_q<scalar_t><<<blocks_q, threads>>>(
	grad_Y.data<scalar_t>(),
	head_q.data<scalar_t>(),
	head_k.data<scalar_t>(),
	paramR.data<scalar_t>(),
	     Y.data<scalar_t>(),
	grad_head_q.data<scalar_t>(),
	grad_head_k.data<scalar_t>(),
	grad_paramR.data<scalar_t>(),
	n_head, bsz, qlen, klen, qleni, kleni, d_head, N_q);
  }));  
  
  return {grad_head_q, grad_head_k, grad_paramR};
}